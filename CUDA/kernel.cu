
#include <hip/hip_runtime.h>

__global__ void kernel(int * vals, int size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	
	for (; tid < size; tid += stride) {
		vals[tid] *= 2;
	}
}

extern "C"
int foo(int size){
	int * vals;
	int * devVals;

	hipHostMalloc((void**)&vals, size * sizeof(vals[0]), hipHostMallocDefault);
	hipMalloc((void**)&devVals, size * sizeof(devVals[0]));

	int i = 0;
	for (; i < size; i++) {
		vals[i] = i;
	}
	hipMemcpy(devVals, vals, size * sizeof(vals[0]), hipMemcpyHostToDevice);
	kernel<<<1, 16, 16>>>(devVals, size);
	hipMemcpy(vals, devVals, size * sizeof(devVals[0]), hipMemcpyDeviceToHost);
	i = 0;
	int tmp = 0;
	for (; i < size; i++) {
		tmp += vals[i];
	}
	return tmp;
}
